#include "hip/hip_runtime.h"
﻿
#include <stdio.h> 
#include <stdlib.h>
#include <time.h>
#include<hip/hip_runtime.h>
#include <>

__global__ void sum(double* d_a, double* d_s)
{
    __shared__ double temp[512];
    int global_index = blockIdx.x * blockDim.x + threadIdx.x;

    temp[threadIdx.x] = d_a[global_index];

    __syncthreads();

    if (threadIdx.x == 0) {
        double summ = 0;
        for (int i = 0; i < blockDim.x; ++i) {
            summ += temp[i];
        }
        atomicAdd(d_s, summ);
    }
}
void cuda_init_device() {
    int dev_count, device = 0;
    hipGetDeviceCount(&dev_count);
    if (dev_count == 0) {
        printf("No cuda device detected \n");
        exit(EXIT_FAILURE);
    }
    for (int device = 0; device < dev_count; ++device) {
        hipDeviceProp_t dev_prop;
        hipGetDeviceProperties(&dev_prop, device);
        if (dev_prop.major < 1 || (dev_prop.major == 1 && dev_prop.minor >= 3)) {
            break;
        }

        if (device == dev_count) {
            printf("No device with 1.3 compute capability was found \n");
            exit(EXIT_FAILURE);
        }
        else {
            hipSetDevice(device);
        }
    }
}
int  main(int argc, char** argv)
{
    long long int i, N = 2097152;

    hipEvent_t start, stop;

    float time_elapsed = 0.0f;
    float gflop = 0.0f;

    if (argc > 1) {
        N = atoi(argv[1]);
    }

    cuda_init_device();

    // Initialize host variables
    double* h_a;
    double h_s = 0;
    h_a = (double*)malloc(N * sizeof(double));

    srand(0);

    for (i = 0; i < N; i++) {  // generate random data
        h_a[i] = (double)rand() / RAND_MAX;
    }

    printf("Running CPU sum for %lld elements\n", N);

    // Initialize device variables
    double* d_a;
    double* d_s;
    hipMalloc((void**)&d_a, sizeof(double) * N);
    hipMalloc((void**)&d_s, sizeof(double));
    hipMemcpy(d_a, h_a, sizeof(double) * N, hipMemcpyHostToDevice);
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Specify amount of block and threads

    int threads = 512;
    int block_amount = (N + threads - 1) / threads;
    hipEventRecord(start, 0);

    sum <<<block_amount, threads >> > (d_a, d_s);  // call compute kernel

    // Copu sum value from device to host memory

    hipMemcpy(&h_s, d_s, sizeof(double), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed, start, stop);

    printf("sum=%.2f\n", h_s);

    gflop = ((N - 1) / time_elapsed / 1E3f);
    printf("sec = %f   GFLOPS = %.3f\n", time_elapsed / 1E3F, gflop);

    hipFree(d_a);
    hipFree(d_s);
    free(h_a);  // free allocated memory
    hipEventDestroy(start);
    hipEventDestroy(stop);
    scanf("%d");
    return EXIT_SUCCESS;
}

